#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iomanip>
#include <iostream>
using std::setw;

const int N = 5;
const int threadsPerBlock = N;
int blocksPerGrid = 1;

// device code
__global__ void prefixSum(float* x, float* c) {
  __shared__ float
      cache[2 * threadsPerBlock];  // declaring a array in shared memory
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIdx = threadIdx.x;  // cacheIdx equal the threadIdx in every block

  if (tid < N) {
    cache[cacheIdx] = x[tid];
  }

  // reduction step: the code below performs iterative scan on cache
  for (int stride = 1; stride <= threadsPerBlock; stride *= 2) {
    __syncthreads();
    int index = (threadIdx.x + 1) * stride * 2 - 1;
    if (index < 2 * threadsPerBlock) {
      cache[index] +=
          cache[index - stride];  // index is alway bigger than stride
    }
    __syncthreads();
  }
  // threadIdx.x+1 = 1,2,3,4....
  // stride index = 1,3,5,7...

  for (int stride = threadsPerBlock / 2; stride > 0; stride /= 2) {
    __syncthreads();
    int index = (threadIdx.x + 1) * stride * 2 - 1;
    if (index < 2 * threadsPerBlock) {
      cache[index] +=
          cache[index - stride];  // index is alway bigger than stride
    }
    __syncthreads();
  }

  // reduction reverse phase
  int stride = 1;
  while (stride <= threadsPerBlock) {
    int index = (threadIdx.x + 1) * stride * 2 - 1;
    if (index < 2 * threadsPerBlock) {
      cache[index + stride] += cache[index];
    }
  }
  __syncthreads();

  if (tid < N) {
    c[tid] = cache[threadIdx.x];
  }
}

// host code
int main() {
  size_t size = N * sizeof(float);
  size_t size_c = blocksPerGrid * sizeof(float);

  // Allocate input vectors h_a and h_b in host memory
  float* h_x = (float*)malloc(size);
  float* h_c = (float*)malloc(size_c);

  // Initialize input vectors
  for (int i = 0; i < N; ++i) {
    h_x[i] = i;
  }

  // Allocate vectors in device memory
  float* d_x;
  float* d_c;

  hipMalloc(&d_x, size);
  hipMalloc(&d_c, size_c);

  // copy vectors from host memory to device memory
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

  // Invoke kernel
  prefixSum<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_c);

  // copy result from device memory to host memory
  // h_c contains the result in host memory
  hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);

  // output the result
  std::cout << "Element" << setw(13) << "Value" << std::endl;
  for (int i = 0; i < N; ++i) {
    std::cout << setw(7) << i << setw(13) << h_c[i] << std::endl;
  }

  // Free device memory
  hipFree(d_x);
  hipFree(d_c);

  // Free host memory
  free(h_x);
  free(h_c);

  hipDeviceReset();
  return 0;
}
