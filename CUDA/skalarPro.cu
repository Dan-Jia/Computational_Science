#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 524288;
const int threadsPerBlock = 1024;
int blocksPerGrid = (N + threadsPerBlock) / threadsPerBlock - 1;

// device code
__global__ void skalarPro(float* a, float* b, float* c) {
  __shared__ float cache[threadsPerBlock];  // declaring a array for every block
                                            // in shared memory

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIdx = threadIdx.x;  // cacheIdx equal the threadIdx in every block

  // save the result of every thread in cache[]
  // if the computing can not once finish
  float temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;  // adding temp per all threads
  }

  cache[cacheIdx] = temp;  // save the result of every thread in cache[cacheIdx]
  __syncthreads();

  // reduce the sum in every cache to cache[0]
  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIdx < i) {
      cache[cacheIdx] += cache[cacheIdx + i];
    }
    __syncthreads();
    i /= 2;
  }

  // store the result of cache[0] into global variable c[]
  if (cacheIdx == 0) {
    c[blockIdx.x] = cache[0];
  }
}

// host code
int main() {
  size_t size = N * sizeof(float);
  size_t size_c = blocksPerGrid * sizeof(float);

  // Allocate input vectors h_a and h_b in host memory
  float* h_a = (float*)malloc(size);
  float* h_b = (float*)malloc(size);
  float* h_c = (float*)malloc(size_c);

  // Initialize input vectors
  for (int i = 0; i < N; i++) {
    h_a[i] = i + 1;
    h_b[i] = N - i;
  }

  // Allocate vectors in device memory
  float* d_a;
  float* d_b;
  float* d_c;

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size_c);

  // copy vectors from host memory to device memory
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // Invoke kernel
  skalarPro<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

  // copy result from device memory to host memory
  // h_c contains the result in host memory
  hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);

  // reduce the sum in every h_c[i] to cache[0]
  int i = blocksPerGrid / 2;
  while (i != 0) {
    for (int j = 0; j < blocksPerGrid; j++) {
      if (j < i) {
        h_c[j] += h_c[j + i];
      }
    }
    i /= 2;
  }

  printf("Das Ergebnis vom Skalarprodukt: %f\n", h_c[0]);

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Free host memory
  free(h_a);
  free(h_b);
  free(h_c);

  hipDeviceReset();
  return 0;
}
